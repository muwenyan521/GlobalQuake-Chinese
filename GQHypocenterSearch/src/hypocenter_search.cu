#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <jni.h>
#include <stdio.h>

#include "globalquake.hpp"
#include "globalquake_jni_GQNativeFunctions.h"
#include "travel_table.hpp"

#define BLOCK_HYPOCS 512
#define BLOCK_REDUCE 256
#define BLOCK_DISTANCES 64

#define STATION_FILEDS 4
#define HYPOCENTER_FILEDS 5

/**
 * STATION:
 * lat | lon | alt | pwave
 * 
 * PRELIMINARY_HYPOCENTER:
 * err | correct (int) | index (int) | origin | depth
 * 
 * RESULT_HYPOCENTER:
 * lat, lon, depth, origin
*/

#define SHARED_TRAVEL_TABLE_SIZE 2048

#define PHI2 2.618033989f
#define PI 3.14159256f

struct depth_profile_t
{
    float depth_resolution;
    float *device_travel_table;
};

bool cuda_initialised = false;
float max_depth_resolution;

int depth_profile_count;
depth_profile_t *depth_profiles = nullptr;
float *f_results_device = nullptr;

size_t total_travel_table_size;

void print_err(const char *msg) {
    hipError_t err = hipGetLastError();
    TRACE(2, "%s 失败: %s (%d)\n", msg, hipGetErrorString(err), err);
}

__host__ void move_on_globe(float from_lat, float from_lon, float angle, float angular_distance, float *lat, float *lon) {
    // calculate angles
    float delta = angular_distance;
    float theta = from_lat;
    float phi = from_lon;
    float gamma = angle;

    // calculate sines and cosines
    float c_theta = cosf(theta);
    float s_theta = sinf(theta);
    float c_phi = cosf(phi);
    float s_phi = sinf(phi);
    float c_delta = cosf(delta);
    float s_delta = sinf(delta);
    float c_gamma = cosf(gamma);
    float s_gamma = sinf(gamma);

    // calculate end vector
    float x = c_delta * c_theta * c_phi - s_delta * (s_theta * c_phi * c_gamma + s_phi * s_gamma);
    float y = c_delta * c_theta * s_phi - s_delta * (s_theta * s_phi * c_gamma - c_phi * s_gamma);
    float z = s_delta * c_theta * c_gamma + c_delta * s_theta;

    // calculate end lat long
    *lat = asinf(z);
    *lon = atan2f(y, x);
}

__device__ void move_on_globe_device(float from_lat, float from_lon, float angle, float angular_distance, float *lat, float *lon) {
    // calculate angles
    float delta = angular_distance;
    float theta = from_lat;
    float phi = from_lon;
    float gamma = angle;

    // calculate sines and cosines
    float c_theta = __cosf(theta);
    float s_theta = __sinf(theta);
    float c_phi = __cosf(phi);
    float s_phi = __sinf(phi);
    float c_delta = __cosf(delta);
    float s_delta = __sinf(delta);
    float c_gamma = __cosf(gamma);
    float s_gamma = __sinf(gamma);

    // calculate end vector
    float x = c_delta * c_theta * c_phi - s_delta * (s_theta * c_phi * c_gamma + s_phi * s_gamma);
    float y = c_delta * c_theta * s_phi - s_delta * (s_theta * s_phi * c_gamma - c_phi * s_gamma);
    float z = s_delta * c_theta * c_gamma + c_delta * s_theta;

    // calculate end lat long
    *lat = asinf(z);
    *lon = atan2f(y, x);
}

// everything is in radians
__device__ float haversine(float lat1, float lon1, float lat2, float lon2) {
    float dlat = lat2 - lat1;
    float dlon = lon2 - lon1;

    // Haversine formula
    float v1 = __sinf(dlat / 2.0f);
    float v2 = __sinf(dlon / 2.0f);
    float a = v1 * v1 + __cosf(lat1) * __cosf(lat2) * v2 * v2;

    float y = __fsqrt_rn(a);
    float x = __fsqrt_rn(1.0f - a);
    float c = atan2f(y, x); // bottleneck of station distances calculation

    return c * 2.0f; // Angular distance in radians
}

// everything in radians
void calculate_params(int points, int index, float max_dist, float from_lat, float from_lon, float *lat, float *lon, float *dist) {
    float ang = (2.0f * PI * (float) index) / PHI2;
    *dist = sqrtf(index) * (max_dist / sqrtf(points - 1.0f));
    move_on_globe(from_lat, from_lon, ang, *dist, lat, lon);
}

__device__ void calculate_params_device(int points, int index, float max_dist, float from_lat, float from_lon, float *lat, float *lon, float *dist) {
    float ang = (2.0f * PI * (float) index) / PHI2;
    *dist = __fsqrt_rn(index) * (max_dist / __fsqrt_rn(points - 1.0f));
    move_on_globe_device(from_lat, from_lon, ang, *dist, lat, lon);
}

__device__ float travel_table_interpolate(float *s_travel_table, float index) {
    // if we are out of bounds, then return some arbitrary value
    if (index >= SHARED_TRAVEL_TABLE_SIZE - 1.0f) {
        return s_travel_table[SHARED_TRAVEL_TABLE_SIZE - 1];
    }

    int index1 = (int) index;
    int index2 = index1 + 1;

    float t = index - index1;
    return (1.0f - t) * s_travel_table[index1] + t * s_travel_table[index2];
}

__device__ inline float *hypocenter_err(float *hypocenter, int grid_size) {
    return &hypocenter[0 * grid_size];
}

__device__ inline int *hypocenter_correct(float *hypocenter, int grid_size) {
    return (int *) &hypocenter[1 * grid_size];
}

__device__ inline float *hypocenter_index(float *hypocenter, int grid_size) {
    return &hypocenter[2 * grid_size];
}

__device__ inline float *hypocenter_origin(float *hypocenter, int grid_size) {
    return &hypocenter[3 * grid_size];
}

__device__ inline float *hypocenter_depth(float *hypocenter, int grid_size) {
    return &hypocenter[4 * grid_size];
}

__device__ inline float heuristic(float correct, float err) {
    return (correct * correct) / (err * err);
}

__device__ void reduce(float *hypocenter_a, float *hypocenter_b, int grid_size) {
    float err_a = *hypocenter_err(hypocenter_a, grid_size);
    float err_b = *hypocenter_err(hypocenter_b, grid_size);

    int correct_a = *hypocenter_correct(hypocenter_a, grid_size);
    int correct_b = *hypocenter_correct(hypocenter_b, grid_size);

    bool swap = heuristic(correct_b, err_b) > heuristic(correct_a, err_a);

    if (swap) {
        *hypocenter_err(hypocenter_a, grid_size) = *hypocenter_err(hypocenter_b, grid_size);
        *hypocenter_correct(hypocenter_a, grid_size) = *hypocenter_correct(hypocenter_b, grid_size);
        *hypocenter_origin(hypocenter_a, grid_size) = *hypocenter_origin(hypocenter_b, grid_size);
        *hypocenter_index(hypocenter_a, grid_size) = *hypocenter_index(hypocenter_b, grid_size);
        *hypocenter_depth(hypocenter_a, grid_size) = *hypocenter_depth(hypocenter_b, grid_size);
    }
}

__global__ void evaluate_hypocenter(float *results,
        float *travel_table,
        float *stations,
        float *station_distances,
        int station_count,
        int points,
        float max_dist,
        float max_depth,
        float p_wave_threshold) {
    extern __shared__ float s_stations[];
    __shared__ float s_travel_table[SHARED_TRAVEL_TABLE_SIZE];
    __shared__ float s_results[BLOCK_HYPOCS * HYPOCENTER_FILEDS];

    int point_index = blockIdx.x * blockDim.x + threadIdx.x;

    float depth = max_depth * (blockIdx.y / (float) (gridDim.y - 1.0f));

    for (int tt_iteration = 0; tt_iteration < ceilf(SHARED_TRAVEL_TABLE_SIZE / static_cast<float>(blockDim.x)); tt_iteration++) {
        int s_index = tt_iteration * blockDim.x + threadIdx.x;
        if (s_index < SHARED_TRAVEL_TABLE_SIZE) {
            s_travel_table[s_index] = travel_table[blockIdx.y * SHARED_TRAVEL_TABLE_SIZE + s_index];
        }
    }

    for (int station_iteration = 0; station_iteration < ceilf(static_cast<float>(station_count * 1) / blockDim.x); station_iteration++) {
        int index = station_iteration * blockDim.x + threadIdx.x;

        if (index < station_count * 1) {
            s_stations[index] = stations[index + 3 * station_count]; // we care only P wave
        }
    }

    __syncthreads();

    if (point_index >= points) {
        return;
    }

    int j = blockIdx.y % station_count;
    float final_origin = 0.0f;

    // trick with changing station that is being used for origin calculation
    {
        float ang_dist = station_distances[point_index + j * points];
        float s_pwave = s_stations[j];
        float expected_travel_time = travel_table_interpolate(s_travel_table, ang_dist);
        float predicted_origin = s_pwave - expected_travel_time;

        final_origin = predicted_origin;
    }

    float err = 0.0;
    int correct = station_count;

    for (int i = 0; i < station_count; i++) {
        float ang_dist = station_distances[point_index + i * points];
        float s_pwave = s_stations[i];
        float expected_travel_time = travel_table_interpolate(s_travel_table, ang_dist);
        float predicted_origin = s_pwave - expected_travel_time;

        float _err = fabsf(predicted_origin - final_origin);

        if (_err > p_wave_threshold) {
            correct--;
            _err = (_err - p_wave_threshold) * 0.1f + p_wave_threshold;
        }

        err += _err * _err;
    }

    s_results[threadIdx.x + blockDim.x * 0] = err;
    *(int *) &s_results[threadIdx.x + blockDim.x * 1] = correct;
    *(int *) (&s_results[threadIdx.x + blockDim.x * 2]) = point_index;
    s_results[threadIdx.x + blockDim.x * 3] = final_origin;
    s_results[threadIdx.x + blockDim.x * 4] = depth;

    __syncthreads();

    // implementation 3 from slides
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s && blockDim.x * blockIdx.x + threadIdx.x + s < points) {
            reduce(&s_results[threadIdx.x], &s_results[threadIdx.x + s], blockDim.x);
            __syncthreads();
        }
    }

    if (threadIdx.x == 0) {
        int idx = blockIdx.y * gridDim.x + blockIdx.x;
        results[idx + 0 * (gridDim.x * gridDim.y)] = s_results[0 * blockDim.x];
        results[idx + 1 * (gridDim.x * gridDim.y)] = s_results[1 * blockDim.x];
        results[idx + 2 * (gridDim.x * gridDim.y)] = s_results[2 * blockDim.x];
        results[idx + 3 * (gridDim.x * gridDim.y)] = s_results[3 * blockDim.x];
        results[idx + 4 * (gridDim.x * gridDim.y)] = s_results[4 * blockDim.x];
    }
}

__global__ void results_reduce(float *out, float *in, int total_size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= total_size) {
        return;
    }
    __shared__ float s_results[HYPOCENTER_FILEDS * BLOCK_REDUCE];

    s_results[threadIdx.x + BLOCK_REDUCE * 0] = in[index + total_size * 0];
    s_results[threadIdx.x + BLOCK_REDUCE * 1] = in[index + total_size * 1];
    s_results[threadIdx.x + BLOCK_REDUCE * 2] = in[index + total_size * 2];
    s_results[threadIdx.x + BLOCK_REDUCE * 3] = in[index + total_size * 3];
    s_results[threadIdx.x + BLOCK_REDUCE * 4] = in[index + total_size * 4];
    __syncthreads();

    // implementation 3 from slides
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s && blockDim.x * blockIdx.x + threadIdx.x + s < total_size) {
            reduce(&s_results[threadIdx.x], &s_results[threadIdx.x + s], blockDim.x);
            __syncthreads();
        }
    }

    if (threadIdx.x == 0) {
        int idx = blockIdx.y * gridDim.x + blockIdx.x;
        out[idx + 0 * (gridDim.x * gridDim.y)] = s_results[0 * blockDim.x];
        out[idx + 1 * (gridDim.x * gridDim.y)] = s_results[1 * blockDim.x];
        out[idx + 2 * (gridDim.x * gridDim.y)] = s_results[2 * blockDim.x];
        out[idx + 3 * (gridDim.x * gridDim.y)] = s_results[3 * blockDim.x];
        out[idx + 4 * (gridDim.x * gridDim.y)] = s_results[4 * blockDim.x];
    }
}

const float ANGLE_TO_INDEX = (SHARED_TRAVEL_TABLE_SIZE - 1.0f) / MAX_ANG;

__global__ void precompute_station_distances(
        float *station_distances, float *stations, int station_count, int points, float max_dist, float from_lat, float from_lon) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= points) {
        return;
    }

    float lat, lon, dist;

    calculate_params_device(points, index, max_dist, from_lat, from_lon, &lat, &lon, &dist);

    for (int i = 0; i < station_count; i++) {
        float s_lat = stations[i + 0 * station_count];
        float s_lon = stations[i + 1 * station_count];
        float ang_dist = haversine(lat, lon, s_lat, s_lon) * 180.0f / PI;  // because travel table is in degrees
        station_distances[index + i * points] = ang_dist * ANGLE_TO_INDEX; // precompute
    }
}

void prepare_travel_table(float *fitted_travel_table, int rows) {
    for (int row = 0; row < rows; row++) {
        for (int column = 0; column < SHARED_TRAVEL_TABLE_SIZE; column++) {
            fitted_travel_table[row * SHARED_TRAVEL_TABLE_SIZE + column] =
                    p_wave_interpolate(column / (SHARED_TRAVEL_TABLE_SIZE - 1.0) * MAX_ANG, (row / (rows - 1.0)) * table_max_depth);
        }
    }
}

// returns (accurately) estimated total GPU memory allocation size given the parameters
size_t get_total_allocation_size(size_t points, size_t station_count, float depth_resolution) {
    size_t result = total_travel_table_size;

    dim3 blocks = { (unsigned int) ceil(static_cast<float>(points) / BLOCK_HYPOCS), (unsigned int) ceil(table_max_depth / depth_resolution) + 1, 1 };

    size_t station_array_size = sizeof(float) * station_count * STATION_FILEDS;
    size_t station_distances_array_size = sizeof(float) * station_count * points;
    size_t results_size = sizeof(float) * HYPOCENTER_FILEDS * (blocks.x * blocks.y * blocks.z);

    size_t temp_results_array_elements = ceil((blocks.x * blocks.y * blocks.z) / static_cast<float>(BLOCK_REDUCE));
    size_t temp_results_array_size = (sizeof(float) * HYPOCENTER_FILEDS * temp_results_array_elements);

    result += station_array_size;
    result += station_distances_array_size;
    result += results_size;
    result += temp_results_array_size;

    return result;
}

JNIEXPORT jlong JNICALL Java_globalquake_jni_GQNativeFunctions_getAllocationSize(JNIEnv *, jclass, jint points, jint stations, jfloat depth_resolution) {
    return get_total_allocation_size(points, stations, depth_resolution);
}

bool run_hypocenter_search(float *stations,
        size_t station_count,
        size_t points,
        int depth_profile_index,
        float max_dist,
        float from_lat,
        float from_lon,
        float *final_result,
        float p_wave_threshold) {
    if (depth_profile_index < 0 || depth_profile_index >= depth_profile_count) {
        TRACE(2, "错误!无效的深度轮廓索引: %d!\n", depth_profile_index);
        return false;
    }

    depth_profile_t *depth_profile = &depth_profiles[depth_profile_index];

    float *device_stations;
    float *device_stations_distances;
    float *device_temp_results;

    if (points < 2) {
        TRACE(2, "错误!至少需要2个点!\n");
        return false;
    }

    if (station_count < 3) {
        TRACE(2, "错误!至少需要3个站点!\n");
        return false;
    }

    bool success = true;

    dim3 blocks = {
        (unsigned int) ceil(static_cast<float>(points) / BLOCK_HYPOCS), (unsigned int) ceil(table_max_depth / depth_profile->depth_resolution) + 1, 1
    };
    dim3 threads = { BLOCK_HYPOCS, 1, 1 };

    if (blocks.y < 2) {
        TRACE(2, "错误!至少需要2个深度点!\n");
        return false;
    }

    size_t station_array_size = sizeof(float) * station_count * STATION_FILEDS;
    size_t station_distances_array_size = sizeof(float) * station_count * points;
    size_t results_size = sizeof(float) * HYPOCENTER_FILEDS * (blocks.x * blocks.y * blocks.z);

    size_t temp_results_array_elements = ceil((blocks.x * blocks.y * blocks.z) / static_cast<float>(BLOCK_REDUCE));
    size_t current_result_count = blocks.x * blocks.y * blocks.z;

    const int block_count = ceil(static_cast<float>(points) / BLOCK_DISTANCES);

    TRACE(1, "台站数组大小(%ld个台站)%.2f kB\n", station_count, station_array_size / (1024.0));
    TRACE(1, "台站距离数组大小%.2f kB\n", station_distances_array_size / (1024.0));
    TRACE(1, "临时结果数组大小%.2f kB\n", (sizeof(float) * HYPOCENTER_FILEDS * temp_results_array_elements) / (1024.0));
    TRACE(1, "结果数组大小为%.2f MB\n", (results_size / (1024.0 * 1024.0)));

    success &= hipMalloc(&device_stations, station_array_size) == hipSuccess;
    success &= hipMemcpy(device_stations, stations, station_array_size, hipMemcpyHostToDevice) == hipSuccess;
    success &= hipMalloc(&device_stations_distances, station_distances_array_size) == hipSuccess;
    success &= hipMalloc(&device_temp_results, sizeof(float) * HYPOCENTER_FILEDS * temp_results_array_elements) == hipSuccess;
    success &= hipMalloc(&f_results_device, results_size) == hipSuccess;

    if (!success) {
        print_err("震中搜索初始化");
        goto cleanup;
    }

    TRACE(1, "网格大小:%d %d %d\n", blocks.x, blocks.y, blocks.z);
    TRACE(1, "区块大小:%d %d %d\n", threads.x, threads.y, threads.z);
    TRACE(1, "总点数:%lld\n", (((long long) (blocks.x * blocks.y * blocks.z)) * (long long) (threads.x * threads.y * threads.z)));

    if (success) {
        precompute_station_distances<<<block_count, BLOCK_DISTANCES>>>(
                device_stations_distances, device_stations, station_count, points, max_dist, from_lat, from_lon);
    }

    success &= hipDeviceSynchronize() == hipSuccess;

    if (!success) {
        print_err("计算站点范围");
        goto cleanup;
    }

    if (success) {
        evaluate_hypocenter<<<blocks, threads, sizeof(float) * station_count>>>(f_results_device,
                depth_profile->device_travel_table,
                device_stations,
                device_stations_distances,
                station_count,
                points,
                max_dist,
                table_max_depth,
                p_wave_threshold);
    }

    success &= hipDeviceSynchronize() == hipSuccess;

    if (!success) {
        print_err("震中搜索");
        goto cleanup;
    }

    while (success && current_result_count > 1) {
        dim3 blocks_reduce = { (unsigned int) ceil(current_result_count / static_cast<double>(BLOCK_REDUCE)), 1, 1 };
        TRACE(1, "正在减少...从 %ld 到 %d\n", current_result_count, blocks_reduce.x);

        results_reduce<<<blocks_reduce, BLOCK_REDUCE>>>(device_temp_results, f_results_device, current_result_count);
        success &= hipDeviceSynchronize() == hipSuccess;

        if (!success) {
            print_err("减少");
            goto cleanup;
        }

        current_result_count = blocks_reduce.x;

        float local_result[HYPOCENTER_FILEDS];

        if (current_result_count == 1) {
            success &= hipMemcpy(local_result, device_temp_results, HYPOCENTER_FILEDS * sizeof(float), hipMemcpyDeviceToHost) == hipSuccess;

            float lat, lon, u_dist;
            calculate_params(points, *(int *) &local_result[2], max_dist, from_lat, from_lon, &lat, &lon, &u_dist);

            final_result[0] = lat;
            final_result[1] = lon;
            final_result[2] = local_result[4];
            final_result[3] = local_result[3];
        } else {
            success &= hipMemcpy(f_results_device, device_temp_results, current_result_count * HYPOCENTER_FILEDS * sizeof(float), hipMemcpyDeviceToDevice) ==
                    hipSuccess;
        }

        if (!success) {
            print_err("CUDA内存拷贝");
            goto cleanup;
        }
    }

cleanup:

    if (device_stations) {
        success &= hipFree(device_stations) == hipSuccess;
    }
    if (device_stations_distances) {
        success &= hipFree(device_stations_distances) == hipSuccess;
    }
    if (device_temp_results) {
        success &= hipFree(device_temp_results) == hipSuccess;
    }
    if (f_results_device) {
        success &= hipFree(f_results_device) == hipSuccess;
    }

    return success;
}

JNIEXPORT jfloatArray JNICALL Java_globalquake_jni_GQNativeFunctions_findHypocenter(JNIEnv *env,
        jclass,
        jfloatArray stations,
        jfloat from_lat,
        jfloat from_lon,
        jlong points,
        int depth_resolution_profile_id,
        jfloat max_dist,
        jfloat p_wave_threshold) {
    size_t station_count = env->GetArrayLength(stations) / STATION_FILEDS;

    float *stations_array = static_cast<float *>(malloc(sizeof(float) * station_count * STATION_FILEDS));
    if (!stations_array) {
        perror("malloc");
        return nullptr;
    }

    jfloat *elements = env->GetFloatArrayElements(stations, 0);
    for (int i = 0; i < station_count * STATION_FILEDS; i++) {
        stations_array[i] = elements[i];
    }

    env->ReleaseFloatArrayElements(stations, elements, 0);

    float final_result[HYPOCENTER_FILEDS];

    bool success = run_hypocenter_search(
            stations_array, station_count, points, depth_resolution_profile_id, max_dist, from_lat, from_lon, final_result, p_wave_threshold);

    free(stations_array);

    jfloatArray result = nullptr;

    if (success) {
        result = env->NewFloatArray(4);

        if (result != nullptr) {
            env->SetFloatArrayRegion(result, 0, 4, final_result);
        }
    }

    return result;
}

bool init_depth_profiles(float *resols, int count) {
    max_depth_resolution = table_max_depth;
    depth_profile_count = count;

    depth_profiles = static_cast<depth_profile_t *>(malloc(count * sizeof(depth_profile_t)));
    if (depth_profiles == nullptr) {
        perror("malloc");
        return false;
    }

    total_travel_table_size = 0;

    for (int i = 0; i < depth_profile_count; i++) {
        float depth_resolution = resols[i];
        if (depth_resolution < max_depth_resolution) {
            max_depth_resolution = depth_resolution;
        }

        depth_profiles[i].depth_resolution = depth_resolution;

        int rows = (unsigned int) ceil(table_max_depth / depth_resolution) + 1;
        size_t table_size = sizeof(float) * rows * SHARED_TRAVEL_TABLE_SIZE;
        total_travel_table_size += table_size;

        TRACE(1, "创建深度剖面,分辨率为 %.2f 千米(%.2f kB)\n", depth_resolution, table_size / 1024.0);

        // todo fitted array
        if (hipMalloc(&depth_profiles[i].device_travel_table, table_size) != hipSuccess) {
            print_err("CUDA内存分配");
            return false;
        }

        float *fitted_travel_table = static_cast<float *>(malloc(table_size));

        if (fitted_travel_table == nullptr) {
            perror("malloc");
            return false;
        } else {
            prepare_travel_table(fitted_travel_table, rows);
            if (hipMemcpy(depth_profiles[i].device_travel_table, fitted_travel_table, table_size, hipMemcpyHostToDevice) != hipSuccess) {
                print_err("CUDA内存复制");
                free(fitted_travel_table);
                return false;
            }

            free(fitted_travel_table);
            fitted_travel_table = NULL;
        }
    }

    return true;
}

/*
 * Class:     globalquake_jni_GQNativeFunctions
 * Method:    initCUDA
 * Signature: ()Z
 */
JNIEXPORT jboolean JNICALL Java_globalquake_jni_GQNativeFunctions_initCUDA(JNIEnv *env, jclass, jfloatArray depth_profiles_array) {
    bool success = true;

    if (depth_profiles_array != nullptr && depth_profiles == nullptr) {
        int depth_profile_count = env->GetArrayLength(depth_profiles_array);
        jfloat *depth_resolutions_array = env->GetFloatArrayElements(depth_profiles_array, 0);

        float depth_resolutions[depth_profile_count];
        for (int i = 0; i < depth_profile_count; i++) {
            depth_resolutions[i] = depth_resolutions_array[i];
        }

        env->ReleaseFloatArrayElements(depth_profiles_array, depth_resolutions_array, 0);

        success &= init_depth_profiles(depth_resolutions, depth_profile_count);
    }

    cuda_initialised = success;
    return success;
}
